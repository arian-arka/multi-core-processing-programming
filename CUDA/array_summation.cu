#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>
#include <time.h>


__global__ void saxpy(int n, float a, float *x, float *y,float *result)
{
     int t_id = threadIdx.x + blockDim.x * blockIdx.x;
     int stride = blockDim.x * gridDim.x;
     for (int i = t_id; i < n; i += stride)
     {
          result[i] = a * x[i] + y[i];
     }

    // int i = blockIdx.x * blockDim.x + threadIdx.x;
    // if (i < n)
    //     result[i] = a * x[i] + y[i];
}

int main(void)
{
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipDeviceSynchronize();
    int N = 1670000000 / 4;
    //N:1 048 576

    float *x, *y,*result, *d_x, *d_y,*d_result;

    x = (float *)malloc(N * sizeof(float));
    y = (float *)malloc(N * sizeof(float));
    result = (float *)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_result, N * sizeof(float));

    for (int i = 0; i < N; i++)
    {
        x[i] = i;
        y[i] = i;
    }

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_result, result, N * sizeof(float), hipMemcpyHostToDevice);
     printf("start\n");
    hipEventRecord(start);

    // Perform SAXPY on 1M elements
    //saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y,d_result);
    saxpy<<<65535, 1024>>>(N, 2.0f, d_x, d_y,d_result);
    //saxpy<<<100, 100>>>(N, 2.0f, d_x, d_y,d_result);
    // saxpy<<<(N + 255) / 256, 256>>>(N, 2.0f, d_x, d_y,d_result);
    //Single-Precision A·X Plus Y
    //z = ax + y
    hipEventRecord(stop);
     printf("end\n");
    hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    //That is CUDA C in a nutshell. 
    // As you can see, the SAXPY kernel contains the same computation as the sequential C version, 
    //but instead of looping over the N elements, we launch a single thread for each of the N elements, 
    //and each thread computes its array index using blockIdx.x*blockDim.x + threadIdx.x.

    printf("milliseconds: %f\n",milliseconds);
    printf("last: %f\n",result[N-1]);

    // float maxError = 0.0f,tmp;
    // for (int i = 0; i < N; i++)
    // {
    //     tmp = 2.0f * x[i] + y[i];
    //     // printf("%f\n",result[i]);
    //     // printf("%f\n",tmp);
    //     // printf("\n");
    //     maxError = max(maxError, abs(result[i] - tmp));
    // }
    // printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_result);
    free(x);
    free(y);
    free(result);
}
// nvcc array_summation.cu -o run -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\BuildTools\VC\Tools\MSVC\14.29.30133\bin\Hostx64\x64"