
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main() {
    print_kernel<<<2, 2>>>();
    hipDeviceSynchronize();
}

//nvcc hello.cu -o hello -ccbin "C:\Program Files (x86)\Microsoft Visual Studio\2019\BuildTools\VC\Tools\MSVC\14.29.30133\bin\Hostx64\x64"